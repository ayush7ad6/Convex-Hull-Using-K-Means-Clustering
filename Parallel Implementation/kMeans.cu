#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>


#define K 3
#define TPB 64
#define MAX_ITER 30

inline __device__ float distance(int x1, int y1, int x2, int y2)
{
	return sqrt((float)(x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1));
}

inline __global__ void kMeansClusterAssignment(int N, int* d_datapoint_x, int* d_datapoint_y, int* d_clust_assn, int* d_centroids_x, int* d_centroids_y)
{

	//get idx for this datapoint
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//bounds check
	if (idx >= N) return;

	//find the closest centroid to this datapoint
	float min_dist = INFINITY;
	int closest_centroid = 0;

	for (int c = 0; c < K; ++c)
	{
		float dist = distance(d_datapoint_x[idx], d_datapoint_y[idx], d_centroids_x[c], d_centroids_y[c]);
		// printf("%f \n", dist);
		if (dist < min_dist)
		{
			min_dist = dist;
			closest_centroid = c;
		}
	}

	//assign closest cluster id for this datapoint/thread
	d_clust_assn[idx] = closest_centroid;
}


inline __global__ void kMeansCentroidUpdate(int N, int* d_datapoints_x, int* d_datapoints_y, int* d_clust_assn, int* d_centroids_x, int* d_centroids_y, int* d_clust_sizes)
{
	//get idx of thread at grid level
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//bounds check
	if (idx >= N) return;

	//get idx of thread at the block level
	const int s_idx = threadIdx.x;

	//put the datapoints and corresponding cluster assignments in shared memory so that they can be summed by thread 0 later
	__shared__ int s_datapoints_x[TPB];
	__shared__ int s_datapoints_y[TPB];

	s_datapoints_x[s_idx] = d_datapoints_x[idx];
	s_datapoints_y[s_idx] = d_datapoints_y[idx];


	__shared__ int s_clust_assn[TPB];
	s_clust_assn[s_idx] = d_clust_assn[idx];

	__syncthreads();

	//it is the thread with idx 0 (in each block) that sums up all the values within the shared array for the block it is in
	if (s_idx == 0)
	{
		int b_clust_datapoint_sums_x[K] = { 0 };
		int b_clust_datapoint_sums_y[K] = { 0 };


		int b_clust_sizes[K] = { 0 };

		for (int j = 0; j < blockDim.x; ++j)
		{
			int clust_id = s_clust_assn[j];
			b_clust_datapoint_sums_x[clust_id] += s_datapoints_x[j];
			b_clust_datapoint_sums_y[clust_id] += s_datapoints_y[j];

			b_clust_sizes[clust_id] += 1;
		}

		//Now we add the sums to the global centroids and add the counts to the global counts.
		for (int z = 0; z < K; ++z) {

			atomicAdd(&d_centroids_x[z], b_clust_datapoint_sums_x[z]);
			atomicAdd(&d_centroids_y[z], b_clust_datapoint_sums_y[z]);

			atomicAdd(&d_clust_sizes[z], b_clust_sizes[z]);
		}
	}

	__syncthreads();

	//currently centroids are just sums, so divide by size to get actual centroids
	if (idx < K) {
		d_centroids_x[idx] = d_centroids_x[idx] / d_clust_sizes[idx];
		d_centroids_y[idx] = d_centroids_y[idx] / d_clust_sizes[idx];
	}
}


inline void find_clusters(int N, int* h_datapoints_x, int* h_datapoints_y, int* h_clust_assign)
{

	//allocate memory on the device for the data points
	int* d_datapoint_x = 0;
	int* d_datapoint_y = 0;
	//allocate memory on the device for the cluster assignments
	int* d_clust_assn = 0;
	//allocate memory on the device for the cluster centroids
	int* d_centroids_x = 0;
	int* d_centroids_y = 0;
	//allocate memory on the device for the cluster sizes
	int* d_clust_sizes = 0;


	hipMalloc(&d_datapoint_x, N * sizeof(int));
	hipMalloc(&d_datapoint_y, N * sizeof(int));
	hipMalloc(&d_clust_assn, N * sizeof(int));
	hipMalloc(&d_centroids_x, K * sizeof(int));
	hipMalloc(&d_centroids_y, K * sizeof(int));

	hipMalloc(&d_clust_sizes, K * sizeof(int));


	int* h_centroids_x = (int*)malloc(K * sizeof(int));
	int* h_centroids_y = (int*)malloc(K * sizeof(int));

	/*int* h_datapoints_x = (int*)malloc(N * sizeof(int));
	int* h_datapoints_y = (int*)malloc(N * sizeof(int));*/

	int* h_clust_sizes = (int*)malloc(K * sizeof(int));
	// int* h_assign = (int*)malloc(N * sizeof(int));

	//srand(time(0));

	//initialize centroids
	/*for (int d = 0; d < N; ++d) {
		h_datapoints_x[d] = rand() % 100;
		h_datapoints_y[d] = rand() % 100;
	}*/

	//Initializing centroids 
	for (int c = 0; c < K; ++c) {
		h_centroids_x[c] = h_datapoints_x[c];
		h_centroids_y[c] = h_datapoints_y[c];
	}

	/*for (int c = 0; c < K; ++c) {
		printf("%d ", h_centroids_x[c]);
	}*/

	hipMemcpy(d_centroids_x, h_centroids_x, K * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_centroids_y, h_centroids_y, K * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_datapoint_x, h_datapoints_y, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_datapoint_y, h_datapoints_y, N * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_clust_sizes, h_clust_sizes, K * sizeof(int), hipMemcpyHostToDevice);

	int cur_iter = 1;

	while (cur_iter < MAX_ITER)
	{
		//call cluster assignment kernel
		// ho gaya 
		kMeansClusterAssignment << <(N + TPB - 1) / TPB, TPB >> > (N, d_datapoint_x, d_datapoint_y, d_clust_assn, d_centroids_x, d_centroids_y);

		//copy new centroids back to host 
		hipMemcpy(h_centroids_x, d_centroids_x, K * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(h_centroids_y, d_centroids_y, K * sizeof(int), hipMemcpyDeviceToHost);

		/*for (int i = 0; i < K; ++i) {
			printf("Iteration %d: centroid %d: %d %d\n", cur_iter, i, h_centroids_x[i], h_centroids_y[i]);
		}*/

		//reset centroids and cluster sizes (will be updated in the next kernel)
		hipMemset(d_centroids_x, 0, K * sizeof(int));
		hipMemset(d_centroids_y, 0, K * sizeof(int));

		hipMemset(d_clust_sizes, 0, K * sizeof(int));

		//call centroid update kernel
		kMeansCentroidUpdate << <(N + TPB - 1) / TPB, TPB >> > (N, d_datapoint_x, d_datapoint_y, d_clust_assn, d_centroids_x, d_centroids_y, d_clust_sizes);

		cur_iter += 1;
	}
	hipMemcpy(h_clust_assign, d_clust_assn, N * sizeof(int), hipMemcpyDeviceToHost);



	/*for (int k = 0; k < K; k++) {
		printf("\ncluster %d \n", k);
		for (int i = 0; i < N; i++) {
			if (h_clust_assign[i] == k) {
				printf("(%d,%d)-%d ", h_datapoints_x[i], h_datapoints_y[i], h_clust_assign[i]);
			}
		}
	}*/

	hipFree(d_datapoint_x);
	hipFree(d_datapoint_y);

	hipFree(d_clust_assn);
	hipFree(d_centroids_x);
	hipFree(d_centroids_y);
	hipFree(d_clust_sizes);

	free(h_centroids_x);
	free(h_centroids_y);

	/*free(h_datapoints_x);
	free(h_datapoints_y);*/

	free(h_clust_sizes);
	// free(h_assign);

	return;
}