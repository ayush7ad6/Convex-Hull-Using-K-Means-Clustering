#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <omp.h>

#include "kMeans.cu"

using namespace std;

int findSide(int p1x, int p1y, int p2x, int p2y, int px, int py)
{
    int val = (py - p1y) * (p2x - p1x) -
        (p2y - p1y) * (px - p1x);

    if (val > 0)
        return 1;
    if (val < 0)
        return -1;
    return 0;
}

// returns a value proportional to the distance
// between the point p and the line joining the
// points p1 and p2
int lineDist(int px, int py, int qx, int qy, int rx, int ry)
{
    return abs((ry - py) * (qx - px) - (qy - py) * (rx - px));
}

int setfunction(int hullx[], int hully[], int index, int convx[], int convy[]) {
    int ind = 0, temp;
    for (int i = 0; i < index - 1; i++) {
        for (int j = 0; j < index - i - 1; j++) {
            if (hullx[j] > hullx[j + 1]) {
                temp = hullx[j];
                hullx[j] = hullx[j + 1];
                hullx[j + 1] = temp;

                temp = hully[j];
                hully[j] = hully[j + 1];
                hully[j + 1] = temp;
            }
            else if (hullx[j] == hullx[j + 1]) {
                if (hully[j] > hully[j + 1]) {
                    temp = hully[j];
                    hully[j] = hully[j + 1];
                    hully[j + 1] = temp;
                }
            }
        }
    }
    int i = 0;
    while (i < index) {
        convx[ind] = hullx[i];
        convy[ind] = hully[i];
        ind++;
        if (hullx[i] == hullx[i + 1] && hully[i] == hully[i + 1])
            i = i + 2;
        else
            i++;
    }
    return ind;
}

// End points of line L are p1 and p2.  side can have value
// 1 or -1 specifying each of the parts made by the line L
int quickHull(int x[], int y[], int n, int px, int py, int qx, int qy, int side, int hullx[], int hully[], int index)
{
    int ind = -1;
    int max_dist = 0;

    // finding the point with maximum distance
    // from L and also on the specified side of L.
    for (int i = 0; i < n; i++)
    {
        int temp = lineDist(px, py, qx, qy, x[i], y[i]);
        if (findSide(px, py, qx, qy, x[i], y[i]) == side && temp > max_dist)
        {
            ind = i;
            max_dist = temp;
        }
    }

    // If no point is found, add the end points
    // of L to the convex hull.
    if (ind == -1)
    {
        hullx[index] = px;
        hully[index] = py;
        index = index + 1;
        hullx[index] = qx;
        hully[index] = qy;
        index = index + 1;
        return index;
    }

    // Recur for the two parts divided by a[ind]
    index = quickHull(x, y, n, x[ind], y[ind], px, py, - findSide(x[ind], y[ind], px, py, qx, qy), hullx, hully, index);
    index = quickHull(x, y, n, x[ind], y[ind], qx, qy, - findSide(x[ind], y[ind], qx, qy, px, py), hullx, hully, index);
    return index;
}

void findHull(int x[], int y[], int n, int* convx, int* convy, int* n_c)
{
    printf("\n\nCluster Size %d;\n", n);
    int* hullx = new int[10000000];
    int* hully = new int[10000000];


    // int hullx[10000000], hully[10000000];
    int index = 0;

    if (n < 3)
    {
        printf("Convex hull not possible\n");
        return;
    }


    int min_x = 0, max_x = 0;
    for (int i = 1; i < n; i++)
    {
        if (x[i] < x[min_x])
            min_x = i;
        if (x[i] > x[max_x])
            max_x = i;
    }

    index = quickHull(x, y, n, x[min_x], y[min_x], x[max_x], y[max_x], 1, hullx, hully, index);


    index = quickHull(x, y, n, x[min_x], y[min_x], x[max_x], y[max_x], -1, hullx, hully, index);

    
    int ind = setfunction(hullx, hully, index, convx, convy);
    *n_c = ind;
    

}



int main()
{
    //Read the 2d points from a file
    const int n = 5000, k = 3;
    double t1, t2, t3, t4;
    int* x = (int*)malloc(n * sizeof(int));
    int* y = (int*)malloc(n * sizeof(int));
    int* convx = (int*)malloc(n * sizeof(int));
    int* convy = (int*)malloc(n * sizeof(int));
    int* kc = (int*)malloc(n * sizeof(int));
    // int x[n], y[n], convx[n], convy[n], kc[n];
    srand(time(0));
    cout << "Generating Random Numbers...!!!" << endl;
    t1 = omp_get_wtime();
    for (int i = 0; i < n; i++) {
        x[i] = rand() % n;
        y[i] = rand() % n;
    }

    //call the clusters' function and get the clusters
    cout << "Obtaining the clusters...!!" << endl;

    t3 = omp_get_wtime();
    
    find_clusters(n, x, y, kc);

    t4 = omp_get_wtime();

    printf("\nTime Taken for K-Means Cluster :%lf \n", t4 - t3);

    /*for (int i = 0; i < n; i++) {
        cout << kc[i] << ' ';
    }*/
    auto xc = new int[k][n];
    auto yc = new int[k][n];
    int clust_size[k];

    // int yc[k][n], xc[k][n], clust_size[k];

    for (int p = 0; p < k; p++) {
        clust_size[p] = 0;
    }
 
    for (int j = 0; j < k; j++) {
        //printf("Cluster %d;\n", j);
        for (int i = 0; i < n; i++) {
            if (kc[i] == j) {
                //printf("point %d: (%d,%d)\n", i, x[i], y[i]);
                clust_size[j] += 1;
                xc[j][clust_size[j] - 1] = x[i];
                yc[j][clust_size[j] - 1] = y[i];
                //cout<<x[i]<<","<<y[i]<<endl;
            }
        }
    }

    for (int i = 0; i < k; i++) {
        printf("clust_size : %d ", clust_size[i]);
    }

    cout << "Finding the convex hulls for each cluster obtained..!!" << endl;
    //for each cluster find the convex hull and store the points

    /*int* con_x = new int[n];
    int* con_y = new int[n];
    int* fin_x = new int[n];
    int* fin_y = new int[n];*/

    int con_x[n], con_y[n], con_n = 0, fin_x[n], fin_y[n], fin_n = 0;
    

    for (int j = 0; j < k; j++) {
        printf("\n\nCluster %d;\n", j);
        findHull(xc[j], yc[j], clust_size[j], con_x, con_y, &con_n);
        for (int i = 0; i < con_n; i++) {
            fin_x[fin_n + i] = con_x[i];
            fin_y[fin_n + i] = con_y[i];
            printf("(%d,%d)\n", con_x[i], con_y[i]);
        }
        fin_n += con_n;
        printf("l_n=%d,g_n=%d\n", con_n, fin_n);
    }

    //now find the convex hull for the points stored above
    printf("\n\nFinding the final Hull..!!\n");

    for (int i = 0; i < fin_n; i++) {
        printf("(%d,%d)\n", fin_x[i], fin_y[i]);
    }
    printf("%d\n", fin_n);

    findHull(fin_x, fin_y, fin_n, con_x, con_y, &con_n);

    printf("***\n");

    for (int i = 0; i < con_n; i++) {
        printf("(%d,%d)\n", con_x[i], con_y[i]);
    }
    t2 = omp_get_wtime();
    printf("Time Taken:%lf\n", t2 - t1);
    return 0;
}